#include "hip/hip_runtime.h"
#include <tcr_cuda.h>
#include <GIRLogger.h>
#include <hipfft/hipfft.h>

#define CUDA_THREADS_PER_BLOCK 256

bool CheckCUDAError( char *tag ) {
	hipError_t error = hipGetLastError();
	if( error != hipSuccess) {
		GIRLogger::LogError( "TCR CUDA error, %s: %s!\n", tag, hipGetErrorString( error ) );
		return false;
	}
	return true;
}

// uses regular dims
__global__ void CUDA_KernelZeroGradient( float* d_gradient ) {
	int i = blockIdx.y*(gridDim.x*blockDim.x) + blockIdx.x*(blockDim.x) + threadIdx.x;
	d_gradient[2*i] = 0;
	d_gradient[2*i+1] = 0;
}

// uses regular dims
__global__ void CUDA_KernelAccumulateGradient( float* d_gradient, float* d_gradient_ch, int num_channels ) {
	int i = blockIdx.y*(gridDim.x*blockDim.x) + blockIdx.x*(blockDim.x) + threadIdx.x;
	for( int channel = 0; channel < num_channels; channel++ ) {
		int ch_i = (blockIdx.y*num_channels + channel)*(gridDim.x*blockDim.x) + blockIdx.x*(blockDim.x) + threadIdx.x;

		d_gradient[2*i] += d_gradient_ch[2*ch_i] / num_channels;
		d_gradient[2*i+1] += d_gradient_ch[2*ch_i+1] / num_channels;
	}
}

// uses regular dims
__global__ void CUDA_KernelCalcTemporalGradient( float* d_gradient, float* d_estimate, int phase_size, int num_phases, float beta ) {
	//beta = 1;
	int i = blockIdx.y*(gridDim.x*blockDim.x) + blockIdx.x*(blockDim.x) + threadIdx.x;

	int phase = blockIdx.y/phase_size;
	int next_phase = (phase+1) % num_phases;
	next_phase = (next_phase*phase_size)*(gridDim.x*blockDim.x) + blockIdx.x*(blockDim.x) + threadIdx.x;
	
	int prev_phase = (phase+num_phases-1) % num_phases;
	prev_phase = (prev_phase*phase_size)*(gridDim.x*blockDim.x) + blockIdx.x*(blockDim.x) + threadIdx.x;
	float beta_squared = 0.00001;
	
	float grad1_real = d_estimate[2*i] - d_estimate[2*next_phase];
	float grad1_imag = d_estimate[2*i+1] - d_estimate[2*next_phase+1];
	float grad1_squared = grad1_real*grad1_real + grad1_imag*grad1_imag;
	
	float grad2_real = -d_estimate[2*i] + d_estimate[2*prev_phase];
	float grad2_imag = -d_estimate[2*i+1] + d_estimate[2*prev_phase+1];
	float grad2_squared = grad2_real*grad2_real + grad2_imag*grad2_imag;
	
	grad1_real = grad1_real / sqrt( grad1_squared + beta_squared );
	grad1_imag = grad1_imag / sqrt( grad1_squared + beta_squared );
	
	grad2_real = grad2_real / sqrt( grad2_squared + beta_squared );
	grad2_imag = grad2_imag / sqrt( grad2_squared + beta_squared );
	
	d_gradient[2*i] -= ( -grad1_real + grad2_real ) * beta;
	d_gradient[2*i+1] -= ( -grad1_imag + grad2_imag ) * beta;

	/*
	next_phase = (blockIdx.y + phase_size*(next_phase-phase)) * (gridDim.x*blockDim.x) + blockIdx.x*(blockDim.x) + threadIdx.x;
	prev_phase = (blockIdx.y + phase_size*(prev_phase-phase)) * (gridDim.x*blockDim.x) + blockIdx.x*(blockDim.x) + threadIdx.x;
	
	d_gradient[2*i] += (2*d_estimate[2*i] - d_estimate[2*next_phase] - d_estimate[2*prev_phase]) * beta;
	d_gradient[2*i+1] += (2*d_estimate[2*i+1] - d_estimate[2*next_phase+1] - d_estimate[2*prev_phase+1]) * beta;
	*/
}

// uses regular dims
__global__ void CUDA_KernelUpdateEstimate( float* d_gradient, float* d_estimate, float step_size ) {
	int i = blockIdx.y*(gridDim.x*blockDim.x) + blockIdx.x*(blockDim.x) + threadIdx.x;
	d_estimate[2*i] -= step_size * d_gradient[2*i];
	d_estimate[2*i+1] -= step_size * d_gradient[2*i+1];
}

// uses channeled dims
__global__ void CUDA_KernelApplySensitivity( float* d_gradient_ch, float* d_estimate, float* d_sensitivity, int num_channels, int max_index ) {
	//todo: load shared memory
	if( blockIdx.x*(blockDim.x) + threadIdx.x <= max_index ) {
		int i = blockIdx.y*(gridDim.x*blockDim.x) + blockIdx.x*(blockDim.x) + threadIdx.x;
		int est_i = (blockIdx.y/num_channels)*(gridDim.x*blockDim.x) + blockIdx.x*(blockDim.x) + threadIdx.x;
		int sense_i = (blockIdx.y%num_channels)*(gridDim.x*blockDim.x) + blockIdx.x*(blockDim.x) + threadIdx.x;
	
		float coil_real = d_sensitivity[2*sense_i];
		float coil_imag = d_sensitivity[2*sense_i + 1];
	
		float source_real = d_estimate[2*est_i];
		float source_imag = d_estimate[2*est_i + 1];
	
		d_gradient_ch[2*i] = source_real*coil_real - source_imag*coil_imag;
		d_gradient_ch[2*i+1] = source_imag*coil_real + source_real*coil_imag;
	}
}

// uses channeled dims
__global__ void CUDA_KernelApplyInvSensitivity( float* d_gradient_ch, float* d_sensitivity, int num_channels, float alpha, int max_index, float scale_factor ) {
	//todo: load shared memory
	if( blockIdx.x*(blockDim.x) + threadIdx.x <= max_index ) {
		int i = blockIdx.y*(gridDim.x*blockDim.x) + blockIdx.x*(blockDim.x) + threadIdx.x;
		int sense_i = (blockIdx.y%num_channels)*(gridDim.x*blockDim.x) + blockIdx.x*(blockDim.x) + threadIdx.x;
	
		float coil_real = d_sensitivity[2*sense_i];
		float coil_imag = -d_sensitivity[2*sense_i + 1];
	
		float gradient_real = d_gradient_ch[2*i];
		float gradient_imag = d_gradient_ch[2*i+1];
	
		d_gradient_ch[2*i] = alpha * scale_factor * (gradient_real*coil_real - gradient_imag*coil_imag);
		d_gradient_ch[2*i+1] = alpha * scale_factor * (gradient_imag*coil_real + gradient_real*coil_imag);
	}
}

// uses channeled dims
__global__ void CUDA_KernelFidelityDifference( float* d_gradient_ch, float* d_mask, float* d_meas, int max_index ) {
	//todo: load shared memory
	if( blockIdx.x*(blockDim.x) + threadIdx.x <= max_index ) {
		int i = blockIdx.y*(gridDim.x*blockDim.x) + blockIdx.x*(blockDim.x) + threadIdx.x;
		/*
		//int mask_i = blockIdx.x*(blockDim.x) + threadIdx.x;
		d_gradient_ch[2*i] = (d_mask[i] * d_gradient_ch[2*i]) - d_meas[2*i];
		d_gradient_ch[2*i + 1] = (d_mask[i] * d_gradient_ch[2*i + 1]) - d_meas[2*i + 1];
		*/
		//!FIX THIS
		if( fabs(d_meas[2*i]) > 1e-20 || fabs(d_meas[2*i+1]) > 1e-20 ) {
			d_gradient_ch[2*i] = d_gradient_ch[2*i] - d_meas[2*i];
			d_gradient_ch[2*i + 1] = d_gradient_ch[2*i + 1] - d_meas[2*i + 1];
		}
		else {
			d_gradient_ch[2*i] = 0;
			d_gradient_ch[2*i + 1] = 0;
		}
	}
}

//!ONLY HANDLES 1 SLICE!!!
extern "C" bool IterateTCR( float alpha, float beta, float step_size, int iterations, float* estimate, float* gradient_ch, float* gradient, float* sensitivity, float* mask, float* meas, int rows, int cols, int channels, int sets, int phases ) {
	GIRLogger::LogInfo( "GPU : alpha=%f, beta=%f, step_size=%f, iterations=%d\n", alpha, beta, step_size, iterations );

	//hipSetDevice( 1 );
	//CheckCUDAError( "hipSetDevice" );

	// Create a 2D FFT plan.
	hipfftHandle plan;
	hipfftPlan2d(&plan, rows, cols, HIPFFT_C2C);
	if( !CheckCUDAError( "hipfftPlan2d" ) ) return false;

	float *d_gradient;
	float *d_estimate;
	float *d_gradient_ch;
	float *d_sensitivity;
	float *d_mask;
	float *d_meas;

	int meas_size =  sizeof(float)*cols*rows*channels*sets*phases*2;
	int est_size =   sizeof(float)*cols*rows*sets*phases*2;
	//int mask_size =  sizeof(float)*cols*rows*channels*partitions;
	int mask_size =  sizeof(float)*cols*rows;
	int sense_size = sizeof(float)*cols*rows*channels*2;

	// allocate memory on device
	hipMalloc( (void**)&d_gradient,    est_size );
	hipMalloc( (void**)&d_estimate,    est_size );
	hipMalloc( (void**)&d_gradient_ch, meas_size );
	hipMalloc( (void**)&d_sensitivity, sense_size );
	hipMalloc( (void**)&d_mask,        mask_size );
	hipMalloc( (void**)&d_meas,        meas_size );
	if( !CheckCUDAError( "cudaMallocs" ) ) return false;

	// copy to device
	hipMemcpy( d_estimate,    estimate,    est_size,   hipMemcpyHostToDevice ); 
	hipMemcpy( d_sensitivity, sensitivity, sense_size, hipMemcpyHostToDevice ); 
	hipMemcpy( d_mask,        mask,        mask_size,  hipMemcpyHostToDevice ); 
	hipMemcpy( d_meas,        meas,        meas_size,  hipMemcpyHostToDevice ); 
	if( !CheckCUDAError( "cudaMemcpys host->device" ) ) return false;

	// set block/grid dimensions
	int num_blocks = (int)ceil( rows*cols / CUDA_THREADS_PER_BLOCK );
	int max_index = rows*cols - 1;
	dim3 ch_dim_block( CUDA_THREADS_PER_BLOCK );
	dim3 ch_dim_grid( num_blocks, channels*sets*phases);

	dim3 dim_block( CUDA_THREADS_PER_BLOCK );
	dim3 dim_grid( num_blocks, sets*phases);

	int num_items = rows * cols;
	float scale_factor = 1.0 / num_items;

	// iterate
	for( int i = 0; i < iterations; i++ ) {
		GIRLogger::LogInfo( "iteration: %d\n", i+1 );

		// launch kernel to reset gradient to 0
		CUDA_KernelZeroGradient<<< dim_grid, dim_block >>>( d_gradient );
		hipDeviceSynchronize();
		if( !CheckCUDAError( "CUDA_KernelZeroGradient" ) ) return false;

		if( alpha > 1e-20 ) {
			// launch apply sensitivity kernel
			CUDA_KernelApplySensitivity<<< ch_dim_grid, ch_dim_block >>>( d_gradient_ch, d_estimate, d_sensitivity, channels, max_index );
			hipDeviceSynchronize();
			if( !CheckCUDAError( "CUDA_KernelApplySensitivity 1" ) ) return false;

			// launch cufft
			int image;
			for( image = 0; image < channels*sets*phases; image++ ) {
				int offset = image*rows*cols*2;
				hipfftExecC2C( plan, (hipfftComplex*)(d_gradient_ch+offset), (hipfftComplex*)(d_gradient_ch+offset), HIPFFT_FORWARD );
			}
			hipDeviceSynchronize();
			if( !CheckCUDAError( "hipfftExecC2C 1" ) ) return false;

			// launch difference kernel
			CUDA_KernelFidelityDifference<<< ch_dim_grid, ch_dim_block >>>( d_gradient_ch, d_mask, d_meas, max_index );
			hipDeviceSynchronize();
			if( !CheckCUDAError( "CUDA_KernelFidelityDifference" ) ) return false;

			// launch inverse cufft
			for( image = 0; image < channels*sets*phases; image++ ) {
				int offset = image*rows*cols*2;
				hipfftExecC2C( plan, (hipfftComplex*)(d_gradient_ch+offset), (hipfftComplex*)(d_gradient_ch+offset), HIPFFT_BACKWARD );
			}
			hipDeviceSynchronize();
			if( !CheckCUDAError( "hipfftExecC2C 2" ) ) return false;
	
			// launch inverse apply sensitivity kernel and scale for IFFT
			CUDA_KernelApplyInvSensitivity<<< ch_dim_grid, ch_dim_block >>>( d_gradient_ch, d_sensitivity, channels, alpha, max_index, scale_factor );
			hipDeviceSynchronize();
			if( !CheckCUDAError( "CUDA_KernelApplySensitivity 2" ) ) return false;

			// launch kernel to accumulate gradient
			CUDA_KernelAccumulateGradient<<< dim_grid, dim_block >>>( d_gradient, d_gradient_ch, channels);
			hipDeviceSynchronize();
			if( !CheckCUDAError( "CUDA_KernelAccumulateGradient" ) ) return false;
		}

		if( beta > 1e-20 ) {
			// launch kernel to calculate temporal gradient
			CUDA_KernelCalcTemporalGradient<<< dim_grid, dim_block >>>( d_gradient, d_estimate, sets, phases, beta );
			hipDeviceSynchronize();
			if( !CheckCUDAError( "CUDA_KernelCalcTemporalGradient" ) ) return false;
		}

		// launch kernel to update the estimate
		CUDA_KernelUpdateEstimate<<< dim_grid, dim_block >>>( d_gradient, d_estimate, step_size );
		hipDeviceSynchronize();
		if( !CheckCUDAError( "CUDA_KernelUpdateEstimate" ) ) return false;
	}

	// copy back to host
	hipMemcpy( estimate, d_estimate, est_size, hipMemcpyDeviceToHost ); 
	if( !CheckCUDAError( "hipMemcpy host<-device" ) ) return false;
	hipMemcpy( gradient, d_gradient, est_size, hipMemcpyDeviceToHost ); 
	if( !CheckCUDAError( "hipMemcpy(ch) host<-device" ) ) return false;
	hipMemcpy( gradient_ch, d_gradient_ch, meas_size, hipMemcpyDeviceToHost ); 
	if( !CheckCUDAError( "hipMemcpy(ch) host<-device" ) ) return false;

	// free up device memory
	hipFree( d_gradient );
	hipFree( d_estimate );
	hipFree( d_gradient_ch );
	hipFree( d_sensitivity );
	hipFree( d_mask );
	hipFree( d_meas );
	if( !CheckCUDAError( "cudaFrees" ) ) return false;
	hipfftDestroy( plan );
	if( !CheckCUDAError( "hipfftDestroy" ) ) return false;

	hipDeviceReset();
	return true;
}
